/**
 * --------------------------------------------------------
 * Universidad del Valle de Guatemala
 * CC3056 - Programación de Microprocesadores
 * --------------------------------------------------------
 * --------------------------------------------------------
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
//#include <json-c/json.h>

#define INICIAL = 16680;
int SIZE = 16680;

// GLOBAL: funcion llamada desde el host y ejecutada en el device (kernel)
__global__ void altura( double *a, double *b, double *c, double *z)
{
	int myID = threadIdx.x + blockDim.x * blockIdx.x;
	//if(myID < z[0])
		c[myID] = a[myID] + b[myID];
	
	
}

// GLOBAL: funcion llamada desde el host y ejecutada en el device (kernel)
__global__ void temperatura( double *d, double *e, double *f, double *y)
{
	int myID = threadIdx.x + blockDim.x * blockIdx.x;
	//if(myID < y[0])
		f[myID] = d[myID] + e[myID];
	

}

__global__ void aceleracionx( double *g, double *h, double *i, double *x)
{
	int myID = threadIdx.x + blockDim.x * blockIdx.x;
	//if(myID < x[0])
		i[myID] = h[myID] + g[myID];
	
}


int main(void) 
{


	//ALMACENAR DATOS EN 3 ARRAYS. 1 TEMP, 1 ALTURA, 1 aceleracionX
	double alturaArray[SIZE], temperaturaArray[SIZE], aceleracionArray[SIZE];

	/*FILE *fp;
    int num = 1024;
    char buffer[num];

    struct json_object *parsed_json
    struct json_object *AcelerometroX
    struct json_object *temperaturaAmbiente
    struct json_object *altitude

	size_t ArSize;
	

    fp = fopen("StefanoLuis.json","r");
    fread(buffer, num,1,fp);
    fclose(fp);

    parsed_json = json_tokener_parse(buffer);

    json_object_object_get_ex(parsed_json, "AcelerometroX", &AcelerometroX)
    json_object_object_get_ex(parsed_json, "temperaturaAmbiente", &temperaturaAmbiente)
    json_object_object_get_ex(parsed_json, "altitude", &altitude)

	ArSize = json_object_array_length(altitude);

	for(size_t i=0;i<ArSize;i++) {
		double AcelTemp = json_object_get_double(json_object_array_get_idx(AcelerometroX, i));
		double TempTemp = json_object_get_double(json_object_array_get_idx(temperaturaAmbiente, i));
		double AltTemp = json_object_get_double(json_object_array_get_idx(altitude, i));

		alturaArray[i] = AltTemp;
		temperaturaArray[i] = TempTemp;
		aceleracionArray[i] = TempTemp
	}	*/

	for(int i=0;i<SIZE;i++) {
		alturaArray[i] = 1;
		temperaturaArray[i] = 1;
		aceleracionArray[i] = 1;
	}

	//WHILE HASTA QUE EL LARGO DE CADA ARRAY SEA 1
	while (SIZE > 1){

		//3 STREAMS. CADA STREAM SE ENCARGARÁ DE 1 VARIABLE
		//--- Stream management ---
		//Object creation
		hipStream_t stream1, stream2, stream3;
		//Stream initialization
		hipStreamCreate(&stream1);
		hipStreamCreate(&stream2);
		hipStreamCreate(&stream3);

		double *a1, *b1, *c1, *z1; // host ptrs to stream 1 arrays
		double *a2, *b2, *c2, *z2; // host ptrs to stream 2 arrays
		double *a3, *b3, *c3, *z3; // host ptrs to stream 2 arrays
		
		double *dev_a1, *dev_b1, *dev_c1, *dev_z1; // stream 1 mem ptrs
		double *dev_a2, *dev_b2, *dev_c2, *dev_z2; // stream 2 mem ptrs
		double *dev_a3, *dev_b3, *dev_c3, *dev_z3; // stream 2 mem ptrs


		int newSize = SIZE+1;
		newSize = newSize-1;

		if(newSize%2==0){
			SIZE = SIZE/2;
		}if(newSize%2!=0){
			SIZE = (SIZE+1)/2;
		}
		
		//stream 1 - mem allocation at Global memmory for device and host
		//---- allocation for device, then host memories required for pinned allocation
		hipMalloc( (void**)&dev_a1, SIZE * sizeof(double) );
		hipMalloc( (void**)&dev_b1, SIZE * sizeof(double) );
		hipMalloc( (void**)&dev_c1, SIZE * sizeof(double) );
		hipMalloc( (void**)&dev_z1, SIZE * sizeof(double) );
		
		hipHostAlloc((void**)&a1,SIZE*sizeof(double),hipHostMallocDefault);
		hipHostAlloc((void**)&b1,SIZE*sizeof(double),hipHostMallocDefault);
		hipHostAlloc((void**)&c1,SIZE*sizeof(double),hipHostMallocDefault);
		hipHostAlloc((void**)&z1,SIZE*sizeof(double),hipHostMallocDefault);
		
		//stream 2 - mem allocation at Global memmory for device and host, in order
		hipMalloc( (void**)&dev_a2, SIZE * sizeof(double) );
		hipMalloc( (void**)&dev_b2, SIZE * sizeof(double) );
		hipMalloc( (void**)&dev_c2, SIZE * sizeof(double) );
		hipMalloc( (void**)&dev_z2, SIZE* sizeof(double) );
		
		hipHostAlloc((void**)&a2,SIZE*sizeof(double),hipHostMallocDefault);
		hipHostAlloc((void**)&b2,SIZE*sizeof(double),hipHostMallocDefault);
		hipHostAlloc((void**)&c2,SIZE*sizeof(double),hipHostMallocDefault);
		hipHostAlloc((void**)&z2,SIZE*sizeof(double),hipHostMallocDefault);

		//stream 3 - mem allocation at Global memmory for device and host, in order
		hipMalloc( (void**)&dev_a3, SIZE * sizeof(double) );
		hipMalloc( (void**)&dev_b3, SIZE * sizeof(double) );
		hipMalloc( (void**)&dev_c3, SIZE * sizeof(double) );
		hipMalloc( (void**)&dev_z3, SIZE* sizeof(double) );
		
		hipHostAlloc((void**)&a3,SIZE*sizeof(double),hipHostMallocDefault);
		hipHostAlloc((void**)&b3,SIZE*sizeof(double),hipHostMallocDefault);
		hipHostAlloc((void**)&c3,SIZE*sizeof(double),hipHostMallocDefault);
		hipHostAlloc((void**)&z3,SIZE*sizeof(double),hipHostMallocDefault);


		for(int i=0;i<SIZE;i++) 
		{
			a1[i] = alturaArray[i];
			b1[i] = alturaArray[i+SIZE];
			
			a2[i] = temperaturaArray[i];
			b2[i] = temperaturaArray[i+SIZE];

			a3[i] = aceleracionArray[i];
			b3[i] = aceleracionArray[i+SIZE];

			z1[i] = SIZE;
			z2[i] = SIZE;
			z3[i] = SIZE;
		}


		newSize = newSize + 1;

		if(newSize%2==0){
			b1[SIZE-1] = 0;

			b2[SIZE-1] = 0;

			b3[SIZE-1] = 0;
		}

		/*printf("\n");
		for(int loop = 0; loop < SIZE; loop++)
      		printf("%f ", a1[loop]);
		printf("\n\t");
		for(int loop = 0; loop < SIZE; loop++)
      		printf("%f ", b1[loop]);*/


		int cantHilos = 1024;
		
		if(SIZE<1024){
			cantHilos = SIZE;
		}

		int blocks = (SIZE/cantHilos)+1;	

		//stream 1
		hipMemcpyAsync(dev_a1,a1,SIZE*sizeof(double),hipMemcpyHostToDevice,stream1);
		hipMemcpyAsync(dev_b1,b1,SIZE*sizeof(double),hipMemcpyHostToDevice,stream1);
		hipMemcpyAsync(dev_z1,z1,SIZE*sizeof(double),hipMemcpyHostToDevice,stream1);
		altura<<<blocks,cantHilos,0,stream1>>>(dev_a1,dev_b1,dev_c1,dev_z1);
		hipMemcpyAsync(c1,dev_c1,SIZE*sizeof(double),hipMemcpyDeviceToHost,stream1);

		//stream 2
		hipMemcpyAsync(dev_a2,a2,SIZE*sizeof(double),hipMemcpyHostToDevice,stream2);
		hipMemcpyAsync(dev_b2,b2,SIZE*sizeof(double),hipMemcpyHostToDevice,stream2);
		hipMemcpyAsync(dev_z2,z2,SIZE*sizeof(double),hipMemcpyHostToDevice,stream2);
		temperatura<<<blocks,cantHilos,0,stream2>>>(dev_a2,dev_b2,dev_c2,dev_z2);
		hipMemcpyAsync(c2,dev_c2,SIZE*sizeof(double),hipMemcpyDeviceToHost,stream2);

		//stream 3
		hipMemcpyAsync(dev_a3,a3,SIZE*sizeof(double),hipMemcpyHostToDevice,stream3);
		hipMemcpyAsync(dev_b3,b3,SIZE*sizeof(double),hipMemcpyHostToDevice,stream3);
		hipMemcpyAsync(dev_z3,z3,SIZE*sizeof(double),hipMemcpyHostToDevice,stream3);
		aceleracionx<<<blocks,cantHilos,0,stream3>>>(dev_a3,dev_b3,dev_c3,dev_z3);
		hipMemcpyAsync(c3,dev_c3,SIZE*sizeof(double),hipMemcpyDeviceToHost,stream3);

		hipStreamSynchronize(stream1); // wait for stream1 to finish
		hipStreamSynchronize(stream2); // wait for stream2 to finish
		hipStreamSynchronize(stream3); // wait for stream2 to finish

		for(int i = 0; i<SIZE; i++){
			alturaArray[i] = c1[i];
			temperaturaArray[i] = c2[i];
			aceleracionArray[i] = c3[i];
		}

		//printf("\n\t POST SIZE: %d, A1 %f, B1 %f",SIZE, c1[0], c1[SIZE-1]);
		
		hipStreamDestroy(stream1); 		// because we care
		hipStreamDestroy(stream2); 
		hipStreamDestroy(stream3); 
		
	}

	printf("\nAltura (promedio total): %f",alturaArray[0]);
	printf("\nTemperatura (promedio total): %f",temperaturaArray[0]);
	printf("\nAceleración en Eje X (promedio total): %f",aceleracionArray[0]);

	return 0;
}