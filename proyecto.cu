/**
 * --------------------------------------------------------
 * Universidad del Valle de Guatemala
 * CC3056 - Programación de Microprocesadores
 * --------------------------------------------------------
 * --------------------------------------------------------
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
//#include <json-c/json.h>

#define INICIAL = 16680;
int SIZE = 16680;

// GLOBAL: funcion llamada desde el host y ejecutada en el device (kernel)
__global__ void altura( double *a, double *b, double *c, double *z)
{
	int myID = threadIdx.x + blockDim.x * blockIdx.x;
	//if(myID < z[0])
		c[myID] = a[myID] + b[myID];
	
	
}

// GLOBAL: funcion llamada desde el host y ejecutada en el device (kernel)
__global__ void temperatura( double *d, double *e, double *f, double *y)
{
	int myID = threadIdx.x + blockDim.x * blockIdx.x;
	//if(myID < y[0])
		f[myID] = d[myID] + e[myID];
	

}

__global__ void aceleracionx( double *g, double *h, double *i, double *x)
{
	int myID = threadIdx.x + blockDim.x * blockIdx.x;
	//if(myID < x[0])
		i[myID] = h[myID] + g[myID];
	
}


int main(void) 
{


	//ALMACENAR DATOS EN 3 ARRAYS. 1 TEMP, 1 ALTURA, 1 aceleracionX
	double alturaArray[SIZE], temperaturaArray[SIZE], aceleracionArray[SIZE];

	/*FILE *fp;
    int num = 1024;
    char buffer[num];

    struct json_object *parsed_json
    struct json_object *AcelerometroX
    struct json_object *temperaturaAmbiente
    struct json_object *altitude

	size_t ArSize;
	

    fp = fopen("StefanoLuis.json","r");
    fread(buffer, num,1,fp);
    fclose(fp);

    parsed_json = json_tokener_parse(buffer);

    json_object_object_get_ex(parsed_json, "AcelerometroX", &AcelerometroX)
    json_object_object_get_ex(parsed_json, "temperaturaAmbiente", &temperaturaAmbiente)
    json_object_object_get_ex(parsed_json, "altitude", &altitude)

	ArSize = json_object_array_length(altitude);

	for(size_t i=0;i<ArSize;i++) {
		double AcelTemp = json_object_get_double(json_object_array_get_idx(AcelerometroX, i));
		double TempTemp = json_object_get_double(json_object_array_get_idx(temperaturaAmbiente, i));
		double AltTemp = json_object_get_double(json_object_array_get_idx(altitude, i));

		alturaArray[i] = AltTemp;
		temperaturaArray[i] = TempTemp;
		aceleracionArray[i] = TempTemp
	}	*/

	for(int i=0;i<SIZE;i++) {
		alturaArray[i] = 1;
		temperaturaArray[i] = 1;
		aceleracionArray[i] = 1;
	}

	//WHILE HASTA QUE EL LARGO DE CADA ARRAY SEA 1
	while (SIZE > 1){

		//3 STREAMS. CADA STREAM SE ENCARGARÁ DE 1 VARIABLE
		//--- Stream management ---
		//Object creation
		hipStream_t stream1, stream2, stream3;
		//Stream initialization
		hipStreamCreate(&stream1);
		hipStreamCreate(&stream2);
		hipStreamCreate(&stream3);

		double *a1, *b1, *c1, *z1; // host ptrs to stream 1 arrays
		double *a2, *b2, *c2, *z2; // host ptrs to stream 2 arrays
		double *a3, *b3, *c3, *z3; // host ptrs to stream 2 arrays
		
		double *dev_a1, *dev_b1, *dev_c1, *dev_z1; // stream 1 mem ptrs
		double *dev_a2, *dev_b2, *dev_c2, *dev_z2; // stream 2 mem ptrs
		double *dev_a3, *dev_b3, *dev_c3, *dev_z3; // stream 2 mem ptrs


		int newSize = SIZE+1;
		newSize = newSize-1;

		if(newSize%2==0){
			SIZE = SIZE/2;
		}if(newSize%2!=0){
			SIZE = (SIZE+1)/2;
		}
		
		//stream 1 - mem allocation at Global memmory for device and host
		//---- allocation for device, then host memories required for pinned allocation
		hipMalloc( (void**)&dev_a1, SIZE * sizeof(double) );
		hipMalloc( (void**)&dev_b1, SIZE * sizeof(double) );
		hipMalloc( (void**)&dev_c1, SIZE * sizeof(double) );
		hipMalloc( (void**)&dev_z1, SIZE * sizeof(double) );
		
		hipHostAlloc((void**)&a1,SIZE*sizeof(double),hipHostMallocDefault);
		hipHostAlloc((void**)&b1,SIZE*sizeof(double),hipHostMallocDefault);
		hipHostAlloc((void**)&c1,SIZE*sizeof(double),hipHostMallocDefault);
		hipHostAlloc((void**)&z1,SIZE*sizeof(double),hipHostMallocDefault);
		
		//stream 2 - mem allocation at Global memmory for device and host, in order
		hipMalloc( (void**)&dev_a2, SIZE * sizeof(double) );
		hipMalloc( (void**)&dev_b2, SIZE * sizeof(double) );
		hipMalloc( (void**)&dev_c2, SIZE * sizeof(double) );
		hipMalloc( (void**)&dev_z2, SIZE* sizeof(double) );
		
		hipHostAlloc((void**)&a2,SIZE*sizeof(double),hipHostMallocDefault);
		hipHostAlloc((void**)&b2,SIZE*sizeof(double),hipHostMallocDefault);
		hipHostAlloc((void**)&c2,SIZE*sizeof(double),hipHostMallocDefault);
		hipHostAlloc((void**)&z2,SIZE*sizeof(double),hipHostMallocDefault);

		//stream 3 - mem allocation at Global memmory for device and host, in order
		hipMalloc( (void**)&dev_a3, SIZE * sizeof(double) );
		hipMalloc( (void**)&dev_b3, SIZE * sizeof(double) );
		hipMalloc( (void**)&dev_c3, SIZE * sizeof(double) );
		hipMalloc( (void**)&dev_z3, SIZE* sizeof(double) );
		
		hipHostAlloc((void**)&a3,SIZE*sizeof(double),hipHostMallocDefault);
		hipHostAlloc((void**)&b3,SIZE*sizeof(double),hipHostMallocDefault);
		hipHostAlloc((void**)&c3,SIZE*sizeof(double),hipHostMallocDefault);
		hipHostAlloc((void**)&z3,SIZE*sizeof(double),hipHostMallocDefault);

		if(newSize%2==0){
			newSize = newSize/2;

			for(int i=0;i<newSize;i++) 
			{
				a1[i] = alturaArray[i];
				b1[i] = alturaArray[i+newSize];
				
				a2[i] = temperaturaArray[i];
				b2[i] = temperaturaArray[i+newSize];

				a3[i] = aceleracionArray[i];
				b3[i] = aceleracionArray[i+newSize];

				z1[i] = SIZE;
				z2[i] = SIZE;
				z3[i] = SIZE;
			}

		}else{
			newSize = (newSize+1)/2;

			for(int i=0;i<newSize;i++) 
			{
				a1[i] = alturaArray[i];
				a2[i] = temperaturaArray[i];
				a3[i] = aceleracionArray[i];

				z1[i] = SIZE;
				z2[i] = SIZE;
				z3[i] = SIZE;
				
				if(i == newSize-1){
					b1[i] = 0;	
					b2[i] = 0;
					b3[i] = 0;
				}
				else{
					b1[i] = alturaArray[i+newSize];
					b2[i] = temperaturaArray[i+newSize];
					b3[i] = aceleracionArray[i+newSize];
				}
				
			}
		}

		int cantHilos = 1024;
		

		if(SIZE<1024){
			cantHilos = SIZE;
		}

		int blocks = ceil(SIZE/cantHilos);
		

		//stream 1
		hipMemcpyAsync(dev_a1,a1,SIZE*sizeof(double),hipMemcpyHostToDevice,stream1);
		hipMemcpyAsync(dev_b1,b1,SIZE*sizeof(double),hipMemcpyHostToDevice,stream1);
		hipMemcpyAsync(dev_z1,z1,SIZE*sizeof(double),hipMemcpyHostToDevice,stream1);
		altura<<<blocks,cantHilos,0,stream1>>>(dev_a1,dev_b1,dev_c1,dev_z1);
		hipMemcpyAsync(c1,dev_c1,SIZE*sizeof(double),hipMemcpyDeviceToHost,stream1);

		//stream 2
		hipMemcpyAsync(dev_a2,a2,SIZE*sizeof(double),hipMemcpyHostToDevice,stream2);
		hipMemcpyAsync(dev_b2,b2,SIZE*sizeof(double),hipMemcpyHostToDevice,stream2);
		hipMemcpyAsync(dev_z2,z2,SIZE*sizeof(double),hipMemcpyHostToDevice,stream2);
		temperatura<<<blocks,cantHilos,0,stream2>>>(dev_a2,dev_b2,dev_c2,dev_z2);
		hipMemcpyAsync(c2,dev_c2,SIZE*sizeof(double),hipMemcpyDeviceToHost,stream2);

		//stream 3
		hipMemcpyAsync(dev_a3,a3,SIZE*sizeof(double),hipMemcpyHostToDevice,stream3);
		hipMemcpyAsync(dev_b3,b3,SIZE*sizeof(double),hipMemcpyHostToDevice,stream3);
		hipMemcpyAsync(dev_z3,z3,SIZE*sizeof(double),hipMemcpyHostToDevice,stream3);
		aceleracionx<<<blocks,cantHilos,0,stream3>>>(dev_a3,dev_b3,dev_c3,dev_z3);
		hipMemcpyAsync(c3,dev_c3,SIZE*sizeof(double),hipMemcpyDeviceToHost,stream3);

		hipStreamSynchronize(stream1); // wait for stream1 to finish
		hipStreamSynchronize(stream2); // wait for stream2 to finish
		hipStreamSynchronize(stream3); // wait for stream2 to finish

		for(int i = 0; i<SIZE; i++){
			alturaArray[i] = c1[i];
			temperaturaArray[i] = c2[i];
			aceleracionArray[i] = c3[i];
		}

		
		hipStreamDestroy(stream1); 		// because we care
		hipStreamDestroy(stream2); 
		hipStreamDestroy(stream3); 
		
	}

	printf("\nAltura (promedio total): %f",alturaArray[0]);
	printf("\nTemperatura (promedio total): %f",temperaturaArray[0]);
	printf("\nAceleración en Eje X (promedio total): %f",aceleracionArray[0]);


	
		


	return 0;
}