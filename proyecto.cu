/**
 * --------------------------------------------------------
 * Universidad del Valle de Guatemala
 * CC3056 - Programación de Microprocesadores
 * --------------------------------------------------------
 * --------------------------------------------------------
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
//#include <json-c/json.h>

#define INICIAL = 16680;
#define INICIAL2 = 5560;
int SIZE = 16680;
int SIZE2 = 5560;

// GLOBAL: funcion llamada desde el host y ejecutada en el device (kernel)
__global__ void suma( double *a, double *b, double *c, double *z)
{
	int myID = threadIdx.x + blockDim.x * blockIdx.x;
	//if(myID < z[0])
		c[myID] = a[myID] + b[myID];

}


int main(void) 
{

	//ALMACENAR DATOS EN 3 ARRAYS. 1 TEMP, 1 ALTURA, 1 aceleracionX
	double alturaArray[SIZE], temperaturaArray[SIZE], aceleracionArray[SIZE];

	double alturaArrayONE[SIZE2], alturaArrayTWO[SIZE2], alturaArrayTHREE[SIZE2];
	double temperaturaArrayONE[SIZE2], temperaturaArrayTWO[SIZE2], temperaturaArrayTHREE[SIZE2];
	double aceleracionArrayONE[SIZE2], aceleracionArrayTWO[SIZE2], aceleracionArrayTHREE[SIZE2];

	/*FILE *fp;
    int num = 1024;
    char buffer[num];

    struct json_object *parsed_json
    struct json_object *AcelerometroX
    struct json_object *temperaturaAmbiente
    struct json_object *altitude

	size_t ArSize;
	

    fp = fopen("StefanoLuis.json","r");
    fread(buffer, num,1,fp);
    fclose(fp);

    parsed_json = json_tokener_parse(buffer);

    json_object_object_get_ex(parsed_json, "AcelerometroX", &AcelerometroX)
    json_object_object_get_ex(parsed_json, "temperaturaAmbiente", &temperaturaAmbiente)
    json_object_object_get_ex(parsed_json, "altitude", &altitude)

	ArSize = json_object_array_length(altitude);

	for(size_t i=0;i<ArSize;i++) {
		double AcelTemp = json_object_get_double(json_object_array_get_idx(AcelerometroX, i));
		double TempTemp = json_object_get_double(json_object_array_get_idx(temperaturaAmbiente, i));
		double AltTemp = json_object_get_double(json_object_array_get_idx(altitude, i));

		alturaArray[i] = AltTemp;
		temperaturaArray[i] = TempTemp;
		aceleracionArray[i] = TempTemp
	}	*/

	for(int i=0;i<SIZE;i++) {
		alturaArray[i] = 9;
		temperaturaArray[i] = 9;
		aceleracionArray[i] = 9;

	}

	for(int i=0;i<SIZE2;i++) {
		alturaArrayONE[i] = 3;
		alturaArrayTWO[i] = 3;
		alturaArrayTHREE[i] = 3;
	 	
		temperaturaArrayONE[i] = 3;
		temperaturaArrayTWO[i] = 3;
		temperaturaArrayTHREE[i] = 3;

		aceleracionArrayONE[i] = 3;
		aceleracionArrayTWO[i] = 3;
		aceleracionArrayTHREE[i] = 3;

	}

	//WHILE HASTA QUE EL LARGO DE CADA ARRAY SEA 1
	while (SIZE > 1){

		//3 STREAMS. CADA STREAM SE ENCARGARÁ DE 1 VARIABLE
		//--- Stream management ---
		//Object creation
		hipStream_t stream1, stream2, stream3;
		//Stream initialization
		hipStreamCreate(&stream1);
		hipStreamCreate(&stream2);
		hipStreamCreate(&stream3);

		double *a1, *b1, *c1, *z1; // host ptrs to stream 1 arrays
		double *a2, *b2, *c2, *z2; 
		double *a3, *b3, *c3, *z3; 
		
		double *dev_a1, *dev_b1, *dev_c1, *dev_z1; // stream 1 mem ptrs
		double *dev_a2, *dev_b2, *dev_c2, *dev_z2; // stream 2 mem ptrs
		double *dev_a3, *dev_b3, *dev_c3, *dev_z3; // stream 2 mem ptrs


		int newSize = SIZE+1;
		newSize = newSize-1;

		if(newSize%2==0){
			SIZE = SIZE/2;
		}if(newSize%2!=0){
			SIZE = (SIZE+1)/2;
		}
		
		//stream 1 - mem allocation at Global memmory for device and host
		//---- allocation for device, then host memories required for pinned allocation
		hipMalloc( (void**)&dev_a1, SIZE * sizeof(double) );
		hipMalloc( (void**)&dev_b1, SIZE * sizeof(double) );
		hipMalloc( (void**)&dev_c1, SIZE * sizeof(double) );
		hipMalloc( (void**)&dev_z1, SIZE * sizeof(double) );
		
		hipHostAlloc((void**)&a1,SIZE*sizeof(double),hipHostMallocDefault);
		hipHostAlloc((void**)&b1,SIZE*sizeof(double),hipHostMallocDefault);
		hipHostAlloc((void**)&c1,SIZE*sizeof(double),hipHostMallocDefault);
		hipHostAlloc((void**)&z1,SIZE*sizeof(double),hipHostMallocDefault);
		
		//stream 2 - mem allocation at Global memmory for device and host, in order
		hipMalloc( (void**)&dev_a2, SIZE * sizeof(double) );
		hipMalloc( (void**)&dev_b2, SIZE * sizeof(double) );
		hipMalloc( (void**)&dev_c2, SIZE * sizeof(double) );
		hipMalloc( (void**)&dev_z2, SIZE* sizeof(double) );
		
		hipHostAlloc((void**)&a2,SIZE*sizeof(double),hipHostMallocDefault);
		hipHostAlloc((void**)&b2,SIZE*sizeof(double),hipHostMallocDefault);
		hipHostAlloc((void**)&c2,SIZE*sizeof(double),hipHostMallocDefault);
		hipHostAlloc((void**)&z2,SIZE*sizeof(double),hipHostMallocDefault);

		//stream 3 - mem allocation at Global memmory for device and host, in order
		hipMalloc( (void**)&dev_a3, SIZE * sizeof(double) );
		hipMalloc( (void**)&dev_b3, SIZE * sizeof(double) );
		hipMalloc( (void**)&dev_c3, SIZE * sizeof(double) );
		hipMalloc( (void**)&dev_z3, SIZE* sizeof(double) );
		
		hipHostAlloc((void**)&a3,SIZE*sizeof(double),hipHostMallocDefault);
		hipHostAlloc((void**)&b3,SIZE*sizeof(double),hipHostMallocDefault);
		hipHostAlloc((void**)&c3,SIZE*sizeof(double),hipHostMallocDefault);
		hipHostAlloc((void**)&z3,SIZE*sizeof(double),hipHostMallocDefault);


		for(int i=0;i<SIZE;i++) 
		{
			a1[i] = alturaArray[i];
			b1[i] = alturaArray[i+SIZE];
			
			a2[i] = temperaturaArray[i];
			b2[i] = temperaturaArray[i+SIZE];

			a3[i] = aceleracionArray[i];
			b3[i] = aceleracionArray[i+SIZE];

			z1[i] = SIZE;
			z2[i] = SIZE;
			z3[i] = SIZE;
		}


		newSize = newSize + 1;

		if(newSize%2==0){
			b1[SIZE-1] = 0;

			b2[SIZE-1] = 0;

			b3[SIZE-1] = 0;
		}

		/*printf("\n");
		for(int loop = 0; loop < SIZE; loop++)
      		printf("%f ", a1[loop]);
		printf("\n\t");
		for(int loop = 0; loop < SIZE; loop++)
      		printf("%f ", b1[loop]);*/


		int cantHilos = 1024;
		
		if(SIZE<1024){
			cantHilos = SIZE;
		}

		int blocks = (SIZE/cantHilos)+1;	

		//stream 1
		hipMemcpyAsync(dev_a1,a1,SIZE*sizeof(double),hipMemcpyHostToDevice,stream1);
		hipMemcpyAsync(dev_b1,b1,SIZE*sizeof(double),hipMemcpyHostToDevice,stream1);
		hipMemcpyAsync(dev_z1,z1,SIZE*sizeof(double),hipMemcpyHostToDevice,stream1);
		suma<<<blocks,cantHilos,0,stream1>>>(dev_a1,dev_b1,dev_c1,dev_z1);
		hipMemcpyAsync(c1,dev_c1,SIZE*sizeof(double),hipMemcpyDeviceToHost,stream1);

		//stream 2
		hipMemcpyAsync(dev_a2,a2,SIZE*sizeof(double),hipMemcpyHostToDevice,stream2);
		hipMemcpyAsync(dev_b2,b2,SIZE*sizeof(double),hipMemcpyHostToDevice,stream2);
		hipMemcpyAsync(dev_z2,z2,SIZE*sizeof(double),hipMemcpyHostToDevice,stream2);
		suma<<<blocks,cantHilos,0,stream2>>>(dev_a2,dev_b2,dev_c2,dev_z2);
		hipMemcpyAsync(c2,dev_c2,SIZE*sizeof(double),hipMemcpyDeviceToHost,stream2);

		//stream 3
		hipMemcpyAsync(dev_a3,a3,SIZE*sizeof(double),hipMemcpyHostToDevice,stream3);
		hipMemcpyAsync(dev_b3,b3,SIZE*sizeof(double),hipMemcpyHostToDevice,stream3);
		hipMemcpyAsync(dev_z3,z3,SIZE*sizeof(double),hipMemcpyHostToDevice,stream3);
		suma<<<blocks,cantHilos,0,stream3>>>(dev_a3,dev_b3,dev_c3,dev_z3);
		hipMemcpyAsync(c3,dev_c3,SIZE*sizeof(double),hipMemcpyDeviceToHost,stream3);

		hipStreamSynchronize(stream1); // wait for stream1 to finish
		hipStreamSynchronize(stream2); // wait for stream2 to finish
		hipStreamSynchronize(stream3); // wait for stream2 to finish

		for(int i = 0; i<SIZE; i++){
			alturaArray[i] = c1[i];
			temperaturaArray[i] = c2[i];
			aceleracionArray[i] = c3[i];
		}

		//printf("\n\t POST SIZE: %d, A1 %f, B1 %f",SIZE, c1[0], c1[SIZE-1]);
		
		hipStreamDestroy(stream1); 		// because we care
		hipStreamDestroy(stream2); 
		hipStreamDestroy(stream3); 
		
	}

	printf("\nAltura (promedio total): %f",alturaArray[0]/16680);
	printf("\nTemperatura (promedio total): %f",temperaturaArray[0]/16680);
	printf("\nAceleración en Eje X (promedio total): %f",aceleracionArray[0]/16680);


	//WHILE HASTA QUE EL LARGO DE CADA ARRAY SEA 1
	while (SIZE2 > 1){

		//3 STREAMS. CADA STREAM SE ENCARGARÁ DE 1 VARIABLE
		//--- Stream management ---
		//Object creation
		hipStream_t stream1, stream2, stream3, stream4, stream5, stream6, stream7, stream8, stream9;
		//Stream initialization
		hipStreamCreate(&stream1);
		hipStreamCreate(&stream2);
		hipStreamCreate(&stream3);
		hipStreamCreate(&stream4);
		hipStreamCreate(&stream5);
		hipStreamCreate(&stream6);
		hipStreamCreate(&stream7);
		hipStreamCreate(&stream8);
		hipStreamCreate(&stream9);


		double *a1, *b1, *c1, *z1; // host ptrs to stream 1 arrays
		double *a2, *b2, *c2, *z2; 
		double *a3, *b3, *c3, *z3; 
		double *a4, *b4, *c4, *z4; 
		double *a5, *b5, *c5, *z5; 
		double *a6, *b6, *c6, *z6; 
		double *a7, *b7, *c7, *z7; 
		double *a8, *b8, *c8, *z8; 
		double *a9, *b9, *c9, *z9; 

		
		double *dev_a1, *dev_b1, *dev_c1, *dev_z1; // stream 1 mem ptrs
		double *dev_a2, *dev_b2, *dev_c2, *dev_z2; // stream 2 mem ptrs
		double *dev_a3, *dev_b3, *dev_c3, *dev_z3; 
		double *dev_a4, *dev_b4, *dev_c4, *dev_z4; 
		double *dev_a5, *dev_b5, *dev_c5, *dev_z5; 
		double *dev_a6, *dev_b6, *dev_c6, *dev_z6; 
		double *dev_a7, *dev_b7, *dev_c7, *dev_z7; 
		double *dev_a8, *dev_b8, *dev_c8, *dev_z8; 
		double *dev_a9, *dev_b9, *dev_c9, *dev_z9; 


		int newSIZE2 = SIZE2+1;
		newSIZE2 = newSIZE2-1;

		if(newSIZE2%2==0){
			SIZE2 = SIZE2/2;
		}if(newSIZE2%2!=0){
			SIZE2 = (SIZE2+1)/2;
		}
		
		//stream 1 - mem allocation at Global memmory for device and host
		//---- allocation for device, then host memories required for pinned allocation
		hipMalloc( (void**)&dev_a1, SIZE2 * sizeof(double) );
		hipMalloc( (void**)&dev_b1, SIZE2 * sizeof(double) );
		hipMalloc( (void**)&dev_c1, SIZE2 * sizeof(double) );
		hipMalloc( (void**)&dev_z1, SIZE2 * sizeof(double) );
		
		hipHostAlloc((void**)&a1,SIZE2*sizeof(double),hipHostMallocDefault);
		hipHostAlloc((void**)&b1,SIZE2*sizeof(double),hipHostMallocDefault);
		hipHostAlloc((void**)&c1,SIZE2*sizeof(double),hipHostMallocDefault);
		hipHostAlloc((void**)&z1,SIZE2*sizeof(double),hipHostMallocDefault);
		
		//stream 2 - mem allocation at Global memmory for device and host, in order
		hipMalloc( (void**)&dev_a2, SIZE2 * sizeof(double) );
		hipMalloc( (void**)&dev_b2, SIZE2 * sizeof(double) );
		hipMalloc( (void**)&dev_c2, SIZE2 * sizeof(double) );
		hipMalloc( (void**)&dev_z2, SIZE2* sizeof(double) );
		
		hipHostAlloc((void**)&a2,SIZE2*sizeof(double),hipHostMallocDefault);
		hipHostAlloc((void**)&b2,SIZE2*sizeof(double),hipHostMallocDefault);
		hipHostAlloc((void**)&c2,SIZE2*sizeof(double),hipHostMallocDefault);
		hipHostAlloc((void**)&z2,SIZE2*sizeof(double),hipHostMallocDefault);

		//stream 3 - mem allocation at Global memmory for device and host, in order
		hipMalloc( (void**)&dev_a3, SIZE2 * sizeof(double) );
		hipMalloc( (void**)&dev_b3, SIZE2 * sizeof(double) );
		hipMalloc( (void**)&dev_c3, SIZE2 * sizeof(double) );
		hipMalloc( (void**)&dev_z3, SIZE2* sizeof(double) );
		
		hipHostAlloc((void**)&a3,SIZE2*sizeof(double),hipHostMallocDefault);
		hipHostAlloc((void**)&b3,SIZE2*sizeof(double),hipHostMallocDefault);
		hipHostAlloc((void**)&c3,SIZE2*sizeof(double),hipHostMallocDefault);
		hipHostAlloc((void**)&z3,SIZE2*sizeof(double),hipHostMallocDefault);

		//stream 4 - mem allocation at Global memmory for device and host, in order
		hipMalloc( (void**)&dev_a4, SIZE2 * sizeof(double) );
		hipMalloc( (void**)&dev_b4, SIZE2 * sizeof(double) );
		hipMalloc( (void**)&dev_c4, SIZE2 * sizeof(double) );
		hipMalloc( (void**)&dev_z4, SIZE2* sizeof(double) );
		
		hipHostAlloc((void**)&a4,SIZE2*sizeof(double),hipHostMallocDefault);
		hipHostAlloc((void**)&b4,SIZE2*sizeof(double),hipHostMallocDefault);
		hipHostAlloc((void**)&c4,SIZE2*sizeof(double),hipHostMallocDefault);
		hipHostAlloc((void**)&z4,SIZE2*sizeof(double),hipHostMallocDefault);

		//stream 5 - mem allocation at Global memmory for device and host, in order
		hipMalloc( (void**)&dev_a5, SIZE2 * sizeof(double) );
		hipMalloc( (void**)&dev_b5, SIZE2 * sizeof(double) );
		hipMalloc( (void**)&dev_c5, SIZE2 * sizeof(double) );
		hipMalloc( (void**)&dev_z5, SIZE2* sizeof(double) );
		
		hipHostAlloc((void**)&a5,SIZE2*sizeof(double),hipHostMallocDefault);
		hipHostAlloc((void**)&b5,SIZE2*sizeof(double),hipHostMallocDefault);
		hipHostAlloc((void**)&c5,SIZE2*sizeof(double),hipHostMallocDefault);
		hipHostAlloc((void**)&z5,SIZE2*sizeof(double),hipHostMallocDefault);

		//stream 6 - mem allocation at Global memmory for device and host, in order
		hipMalloc( (void**)&dev_a6, SIZE2 * sizeof(double) );
		hipMalloc( (void**)&dev_b6, SIZE2 * sizeof(double) );
		hipMalloc( (void**)&dev_c6, SIZE2 * sizeof(double) );
		hipMalloc( (void**)&dev_z6, SIZE2* sizeof(double) );
		
		hipHostAlloc((void**)&a6,SIZE2*sizeof(double),hipHostMallocDefault);
		hipHostAlloc((void**)&b6,SIZE2*sizeof(double),hipHostMallocDefault);
		hipHostAlloc((void**)&c6,SIZE2*sizeof(double),hipHostMallocDefault);
		hipHostAlloc((void**)&z6,SIZE2*sizeof(double),hipHostMallocDefault);

		//stream 7 - mem allocation at Global memmory for device and host, in order
		hipMalloc( (void**)&dev_a7, SIZE2 * sizeof(double) );
		hipMalloc( (void**)&dev_b7, SIZE2 * sizeof(double) );
		hipMalloc( (void**)&dev_c7, SIZE2 * sizeof(double) );
		hipMalloc( (void**)&dev_z7, SIZE2* sizeof(double) );
		
		hipHostAlloc((void**)&a7,SIZE2*sizeof(double),hipHostMallocDefault);
		hipHostAlloc((void**)&b7,SIZE2*sizeof(double),hipHostMallocDefault);
		hipHostAlloc((void**)&c7,SIZE2*sizeof(double),hipHostMallocDefault);
		hipHostAlloc((void**)&z7,SIZE2*sizeof(double),hipHostMallocDefault);

		//stream 8 - mem allocation at Global memmory for device and host, in order
		hipMalloc( (void**)&dev_a8, SIZE2 * sizeof(double) );
		hipMalloc( (void**)&dev_b8, SIZE2 * sizeof(double) );
		hipMalloc( (void**)&dev_c8, SIZE2 * sizeof(double) );
		hipMalloc( (void**)&dev_z8, SIZE2* sizeof(double) );
		
		hipHostAlloc((void**)&a8,SIZE2*sizeof(double),hipHostMallocDefault);
		hipHostAlloc((void**)&b8,SIZE2*sizeof(double),hipHostMallocDefault);
		hipHostAlloc((void**)&c8,SIZE2*sizeof(double),hipHostMallocDefault);
		hipHostAlloc((void**)&z8,SIZE2*sizeof(double),hipHostMallocDefault);

		//stream 9 - mem allocation at Global memmory for device and host, in order
		hipMalloc( (void**)&dev_a9, SIZE2 * sizeof(double) );
		hipMalloc( (void**)&dev_b9, SIZE2 * sizeof(double) );
		hipMalloc( (void**)&dev_c9, SIZE2 * sizeof(double) );
		hipMalloc( (void**)&dev_z9, SIZE2* sizeof(double) );
		
		hipHostAlloc((void**)&a9,SIZE2*sizeof(double),hipHostMallocDefault);
		hipHostAlloc((void**)&b9,SIZE2*sizeof(double),hipHostMallocDefault);
		hipHostAlloc((void**)&c9,SIZE2*sizeof(double),hipHostMallocDefault);
		hipHostAlloc((void**)&z9,SIZE2*sizeof(double),hipHostMallocDefault);

		for(int i=0;i<SIZE2;i++) 
		{
			a1[i] = alturaArrayONE[i];
			b1[i] = alturaArrayONE[i+SIZE2];
			a2[i] = alturaArrayTWO[i];
			b2[i] = alturaArrayTWO[i+SIZE2];
			a3[i] = alturaArrayTHREE[i];
			b3[i] = alturaArrayTHREE[i+SIZE2];

			a4[i] = temperaturaArrayONE[i];
			b4[i] = temperaturaArrayONE[i+SIZE2];
			a5[i] = temperaturaArrayTWO[i];
			b5[i] = temperaturaArrayTWO[i+SIZE2];
			a6[i] = temperaturaArrayTHREE[i];
			b6[i] = temperaturaArrayTHREE[i+SIZE2];

			a7[i] = aceleracionArrayONE[i];
			b7[i] = aceleracionArrayONE[i+SIZE2];
			a8[i] = aceleracionArrayTWO[i];
			b8[i] = aceleracionArrayTWO[i+SIZE2];
			a9[i] = aceleracionArrayTHREE[i];
			b9[i] = aceleracionArrayTHREE[i+SIZE2];

			z1[i] = SIZE2;
			z2[i] = SIZE2;
			z3[i] = SIZE2;
			z4[i] = SIZE2;
			z5[i] = SIZE2;
			z6[i] = SIZE2;
			z7[i] = SIZE2;
			z8[i] = SIZE2;
			z9[i] = SIZE2;
		}

		newSIZE2 = newSIZE2 + 1;

		if(newSIZE2%2==0){
			b1[SIZE2-1] = 0;
			b2[SIZE2-1] = 0;
			b3[SIZE2-1] = 0;

			b4[SIZE2-1] = 0;
			b5[SIZE2-1] = 0;
			b6[SIZE2-1] = 0;

			b7[SIZE2-1] = 0;
			b8[SIZE2-1] = 0;
			b9[SIZE2-1] = 0;
		}

		int cantHilos = 1024;
		
		if(SIZE2<1024){
			cantHilos = SIZE2;
		}

		int blocks = (SIZE2/cantHilos)+1;	

		//stream 1
		hipMemcpyAsync(dev_a1,a1,SIZE2*sizeof(double),hipMemcpyHostToDevice,stream1);
		hipMemcpyAsync(dev_b1,b1,SIZE2*sizeof(double),hipMemcpyHostToDevice,stream1);
		hipMemcpyAsync(dev_z1,z1,SIZE2*sizeof(double),hipMemcpyHostToDevice,stream1);
		suma<<<blocks,cantHilos,0,stream1>>>(dev_a1,dev_b1,dev_c1,dev_z1);
		hipMemcpyAsync(c1,dev_c1,SIZE2*sizeof(double),hipMemcpyDeviceToHost,stream1);

		//stream 2
		hipMemcpyAsync(dev_a2,a2,SIZE2*sizeof(double),hipMemcpyHostToDevice,stream2);
		hipMemcpyAsync(dev_b2,b2,SIZE2*sizeof(double),hipMemcpyHostToDevice,stream2);
		hipMemcpyAsync(dev_z2,z2,SIZE2*sizeof(double),hipMemcpyHostToDevice,stream2);
		suma<<<blocks,cantHilos,0,stream2>>>(dev_a2,dev_b2,dev_c2,dev_z2);
		hipMemcpyAsync(c2,dev_c2,SIZE2*sizeof(double),hipMemcpyDeviceToHost,stream2);

		//stream 3
		hipMemcpyAsync(dev_a3,a3,SIZE2*sizeof(double),hipMemcpyHostToDevice,stream3);
		hipMemcpyAsync(dev_b3,b3,SIZE2*sizeof(double),hipMemcpyHostToDevice,stream3);
		hipMemcpyAsync(dev_z3,z3,SIZE2*sizeof(double),hipMemcpyHostToDevice,stream3);
		suma<<<blocks,cantHilos,0,stream3>>>(dev_a3,dev_b3,dev_c3,dev_z3);
		hipMemcpyAsync(c3,dev_c3,SIZE2*sizeof(double),hipMemcpyDeviceToHost,stream3);

		//stream 4
		hipMemcpyAsync(dev_a4,a4,SIZE2*sizeof(double),hipMemcpyHostToDevice,stream4);
		hipMemcpyAsync(dev_b4,b4,SIZE2*sizeof(double),hipMemcpyHostToDevice,stream4);
		hipMemcpyAsync(dev_z4,z4,SIZE2*sizeof(double),hipMemcpyHostToDevice,stream4);
		suma<<<blocks,cantHilos,0,stream4>>>(dev_a4,dev_b4,dev_c4,dev_z4);
		hipMemcpyAsync(c4,dev_c4,SIZE2*sizeof(double),hipMemcpyDeviceToHost,stream4);

		//stream 5
		hipMemcpyAsync(dev_a5,a5,SIZE2*sizeof(double),hipMemcpyHostToDevice,stream5);
		hipMemcpyAsync(dev_b5,b5,SIZE2*sizeof(double),hipMemcpyHostToDevice,stream5);
		hipMemcpyAsync(dev_z5,z5,SIZE2*sizeof(double),hipMemcpyHostToDevice,stream5);
		suma<<<blocks,cantHilos,0,stream5>>>(dev_a5,dev_b5,dev_c5,dev_z5);
		hipMemcpyAsync(c5,dev_c5,SIZE2*sizeof(double),hipMemcpyDeviceToHost,stream5);

		//stream 6
		hipMemcpyAsync(dev_a6,a6,SIZE2*sizeof(double),hipMemcpyHostToDevice,stream6);
		hipMemcpyAsync(dev_b6,b6,SIZE2*sizeof(double),hipMemcpyHostToDevice,stream6);
		hipMemcpyAsync(dev_z6,z6,SIZE2*sizeof(double),hipMemcpyHostToDevice,stream6);
		suma<<<blocks,cantHilos,0,stream6>>>(dev_a6,dev_b6,dev_c6,dev_z6);
		hipMemcpyAsync(c6,dev_c6,SIZE2*sizeof(double),hipMemcpyDeviceToHost,stream6);

		//stream 7
		hipMemcpyAsync(dev_a7,a7,SIZE2*sizeof(double),hipMemcpyHostToDevice,stream7);
		hipMemcpyAsync(dev_b7,b7,SIZE2*sizeof(double),hipMemcpyHostToDevice,stream7);
		hipMemcpyAsync(dev_z7,z7,SIZE2*sizeof(double),hipMemcpyHostToDevice,stream7);
		suma<<<blocks,cantHilos,0,stream7>>>(dev_a7,dev_b7,dev_c7,dev_z7);
		hipMemcpyAsync(c7,dev_c7,SIZE2*sizeof(double),hipMemcpyDeviceToHost,stream7);

		//stream 8
		hipMemcpyAsync(dev_a8,a8,SIZE2*sizeof(double),hipMemcpyHostToDevice,stream8);
		hipMemcpyAsync(dev_b8,b8,SIZE2*sizeof(double),hipMemcpyHostToDevice,stream8);
		hipMemcpyAsync(dev_z8,z8,SIZE2*sizeof(double),hipMemcpyHostToDevice,stream8);
		suma<<<blocks,cantHilos,0,stream8>>>(dev_a8,dev_b8,dev_c8,dev_z8);
		hipMemcpyAsync(c8,dev_c8,SIZE2*sizeof(double),hipMemcpyDeviceToHost,stream8);

		//stream 9
		hipMemcpyAsync(dev_a9,a9,SIZE2*sizeof(double),hipMemcpyHostToDevice,stream9);
		hipMemcpyAsync(dev_b9,b9,SIZE2*sizeof(double),hipMemcpyHostToDevice,stream9);
		hipMemcpyAsync(dev_z9,z9,SIZE2*sizeof(double),hipMemcpyHostToDevice,stream9);
		suma<<<blocks,cantHilos,0,stream9>>>(dev_a9,dev_b9,dev_c9,dev_z9);
		hipMemcpyAsync(c9,dev_c9,SIZE2*sizeof(double),hipMemcpyDeviceToHost,stream9);

		hipStreamSynchronize(stream1); // wait for stream1 to finish
		hipStreamSynchronize(stream2); // wait for stream2 to finish
		hipStreamSynchronize(stream3);
		hipStreamSynchronize(stream4);
		hipStreamSynchronize(stream5);
		hipStreamSynchronize(stream6);
		hipStreamSynchronize(stream7);
		hipStreamSynchronize(stream8);
		hipStreamSynchronize(stream9);

		for(int i = 0; i<SIZE2; i++){
			alturaArrayONE[i] = c1[i];
			alturaArrayTWO[i] = c2[i];
			alturaArrayTHREE[i] = c3[i];
			
			temperaturaArrayONE[i] = c4[i];
			temperaturaArrayTWO[i] = c5[i];
			temperaturaArrayTHREE[i] = c6[i];

			aceleracionArrayONE[i] = c7[i];
			aceleracionArrayTWO[i] = c8[i];
			aceleracionArrayTHREE[i] = c9[i];
		}

		hipStreamDestroy(stream1); 		// because we care
		hipStreamDestroy(stream2); 
		hipStreamDestroy(stream3); 
		hipStreamDestroy(stream4); 
		hipStreamDestroy(stream5); 
		hipStreamDestroy(stream6); 
		hipStreamDestroy(stream7); 
		hipStreamDestroy(stream8);
		hipStreamDestroy(stream9);  
		
	}

	printf("\n\nAltura (promedio DIA 1): %f",alturaArrayONE[0]/5560);
	printf("\nAltura (promedio DIA 2): %f",alturaArrayTWO[0]/5560);
	printf("\nAltura (promedio DIA 3): %f",alturaArrayTHREE[0]/5560);

	printf("\n\nTemperatura (promedio DIA 1): %f",temperaturaArrayONE[0]/5560);
	printf("\nTemperatura (promedio DIA 2): %f",temperaturaArrayTWO[0]/5560);
	printf("\nTemperatura (promedio DIA 3): %f",temperaturaArrayTHREE[0]/5560);

	printf("\n\nAceleración en Eje X (promedio DIA 1): %f",aceleracionArrayONE[0]/5560);
	printf("\nAceleración en Eje X (promedio DIA 2): %f",aceleracionArrayTWO[0]/5560);
	printf("\nAceleración en Eje X (promedio DIA 3): %f",aceleracionArrayTHREE[0]/5560);

	printf("\n\n\nStefano Aragoni, Luis Santos. \n~ se utilizaron 12 streams~.\n");


	return 0;
}